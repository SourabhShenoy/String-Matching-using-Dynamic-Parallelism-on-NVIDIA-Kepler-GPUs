#include "hip/hip_runtime.h"
//regular upto 32, till 256 4 bytes/thread, till 1024 8bytes, above that try 8,12,16
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <fstream>
# include <sys/time.h>
#define ASIZE 256
struct timeval tim;
double dTime1; 

__global__ void compare(int idx,char* x, char* y,int results[],int chunk) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
		switch(chunk) {
			case 16:
				if(x[id*chunk+15]!=y[idx+id*chunk+15] || x[id*chunk+14]!=y[idx+id*chunk+14] || x[id*chunk+13]!=y[idx+id*chunk+13] ||  x[id*chunk+12]!=y[idx+id*chunk+12] || x[id*chunk+11]!=y[idx+id*chunk+11] || x[id*chunk+10]!=y[idx+id*chunk+10] || x[id*chunk+9]!=y[idx+id*chunk+9] ||  x[id*chunk+8]!=y[idx+id*chunk+8]){
						results[idx]=0;
						return;
					}
			case 8:

				if(x[id*chunk+4]!=y[idx+id*chunk+4] || x[id*chunk+5]!=y[idx+id*chunk+5] || x[id*chunk+6]!=y[idx+id*chunk+6] ||  x[id*chunk+7]!=y[idx+id*chunk+7]){
						results[idx]=0;
						return;
					}
	//		case 4:
				if(x[id*chunk+1]!=y[idx+id*chunk+1] || x[id*chunk+2]!=y[idx+id*chunk+2] || x[id*chunk+3]!=y[idx+id*chunk+3]){
						results[idx]=0;
						return;
					}
		//	case 1:
			if(x[id*chunk]!=y[idx+id*chunk]){
						results[idx]=0;
						return;
					} else {
						return;
					}
		}

/*
	if(x[id*4]!=y[idx+id*4] || x[id*4+1]!=y[idx+id*4+1] || x[id*4+2]!=y[idx+id*4+2]|| x[id*4+3]!=y[idx+id*4+3]) {
		results[idx]=0;
		return;
	} else {
		return;
	}
*/
}


__global__ void search(char *x, int m, char* y, int n, int results[],int chunk) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
     
    if ( idx > (n - m) ) {results[idx]=0; return;}

	   if(x[0]==y[idx] && x[m-1]==y[idx+m-1]) {
		compare<<<1,m/chunk>>>(idx,x,y,results,chunk);
		return;
	  } else {
		results[idx]=0;
		return;
	}
}

char* readfile(const char* filename) {
	FILE* f;
	char* data;
	f= fopen(filename, "r");
	if ( f != NULL )  {
		fseek(f,0,SEEK_END);
	    int size=ftell(f);
		fseek(f,0,SEEK_SET);
		data = (char*)malloc((size+1) * sizeof(char));
		fread(data, size,1,f);
	}
	fclose(f);
	return data;
}

void display_results(int n, int  res[]) {
	int c=0;
    for( int i =0; i < n; ++i )
        if ( res[i] == 1 )
		c++;
		printf("\n\nCount:%d\n\n",c);
//            printf("%d. Found match at %d\n",j++, i);

}

int main(int argc, char* argv[]) {
    int cuda_device = 0;
    size_t n = 0;
    size_t m = 0;

    if ( argc < 4 ) {
  //      printf("Usage: ./a.out <device number> <pattern> <data file>\n");
        return -1;
    }

    if( argc > 1 )
        cuda_device = atoi( argv[1] );

	char* mainString = readfile(argv[3]);
    char* subString = (char*) malloc( (strlen(argv[2])) * sizeof(char) );
    strcpy(subString, argv[2]);
    n = strlen(mainString);
    m = strlen(subString);
		int chunk=0;
	int* results=(int*)malloc(n * sizeof(int));
	for( int i = 0; i < n; ++i ) {
		results[i]=1;
	}

  //  hipError_t error;
    hipEvent_t start_event, stop_event;
    float time1;
    
    checkCudaErrors( hipEventCreate(&start_event) );
	checkCudaErrors( hipEventCreate(&stop_event) );

    int num_devices=0;
    checkCudaErrors( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
  //      printf("Your system does not have a CUDA capable device\n");
        return 1;
    }
/*
    if( cuda_device >= num_devices )
    {
		if(num_devices==0)
//			printf("You have only 1 device and it's id is 0\n");
		else    
//		    printf("choose device ID between 0 and %d\n", num_devices-1);
        return 1;
    }
*/
    //hipSetDevice( cuda_device );
    hipDeviceProp_t deviceProp;

    checkCudaErrors( hipGetDeviceProperties(&deviceProp, cuda_device) );
 //   if( (1 == deviceProp.major) && (deviceProp.minor < 1))
//   printf("%s does not have compute capability 1.1 or later\n", deviceProp.name);

//    printf("Device name : %s\n", deviceProp.name );
//    printf("CUDA Capable SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
 //   printf("array_size   = %zd\n", n);

    char* d_substr = 0;
    char* d_text = 0;
    int* d_results = 0;

    checkCudaErrors( hipMalloc((void**)&d_results, n * sizeof(int)) );
    checkCudaErrors( hipMalloc((void**)&d_substr, (m)*sizeof(char)) );
    checkCudaErrors( hipMalloc((void**)&d_text, (strlen(mainString))*sizeof(char)) );
    checkCudaErrors( hipMemcpy(d_results, results, sizeof(int) * n, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy(d_text, mainString, sizeof(char)*(strlen(mainString)), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy(d_substr, subString, sizeof(char)*(m), hipMemcpyHostToDevice) );
//    error = hipGetLastError();
 //   printf("%s\n", hipGetErrorString(error));

    dim3 threadsPerBlocks(ASIZE, 1);
    int t = n / threadsPerBlocks.x;
    int t1 = n % threadsPerBlocks.x;
    if ( t1 != 0 ) t += 1;
    dim3 numBlocks(t,1);

//    printf("Launching kernel with blocks=%d, threadsperblock=%d\n", numBlocks.x, threadsPerBlocks.x);
		if(m<32) {
			chunk=1;
		} else if (m<256) {
			chunk=4;
		} else if (m<1024) {
			chunk=8;
		} else {
			chunk=16;
		}

    hipEventRecord(start_event, 0);
		    search<<<numBlocks,threadsPerBlocks>>>(d_substr, m, d_text, n, d_results, chunk);
    hipDeviceSynchronize();
    hipEventRecord(stop_event, 0);
    hipEventSynchronize( stop_event );
    hipEventElapsedTime( &time1, start_event, stop_event );

    hipEventDestroy( start_event );
    hipEventDestroy( stop_event );

    printf("%lf\t",time1);

    checkCudaErrors( hipMemcpy(results, d_results, n * sizeof(int), hipMemcpyDeviceToHost) );
	display_results(n, results);

	hipFree(d_substr);
	hipFree(d_text);
	hipFree(d_results);
	free(mainString);
	free(subString);
	free(results);
	
	hipDeviceReset();
}
