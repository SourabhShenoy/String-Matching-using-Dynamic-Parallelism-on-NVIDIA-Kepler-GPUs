#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <fstream>

#define ASIZE 256
#define PRIME 1000009

__global__ void processPattern(char* x ,int m, int shifts[]) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if ( idx >= m ) return;    
    char c = x[idx];
    for( int i = m - 1; i >= idx; --i ) {
        if ( x[i] == c ) {
            shifts[c] = m - i;
            return;
        }
    }
}

__global__ void compare(int idx,char *x, char *y, int m, int* results) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
//	printf("%d\t%d\n",idx,id);
	
	if(x[id]!=y[idx+id]) {
		results[idx]=0;
		return;
	} else {
		return;
	}
}


__global__ void search(char *x, int m, char* y, int n, int shifts[], int indx[], int results[]) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if ( idx > (n - m) ) {		        results[idx] = 0; return; }
    if ( indx[idx] != idx ) {		        results[idx] = 0; return; }

	if(x[0]==y[idx] && x[m-1]==y[idx+m-1]) {
/*
if(idx>1000 && idx<1100) {
	compare<<<1,m>>>(idx);
}
*/
	if(m>2)
	compare<<<1,m>>>(idx,x,y,m,results);

/*
		for( int i = 0; i < m; ++i ) {
		    if ( x[i] != y[idx + i] ) {
		        results[idx] = 0;
		        return;
		    }
		}
*/
	} else {
		        results[idx] = 0;
	}
}

char* readfile(const char* filename) {
	FILE* f;
	char* data;
	f= fopen(filename, "r");
	if ( f != NULL )  {
		fseek(f,0,SEEK_END);
	    int size=ftell(f);
		fseek(f,0,SEEK_SET);
		data = (char*)malloc((size) * sizeof(char));
		fread(data, size,1,f);
	}
	fclose(f);
	return data;
}

void precomputeShiftIndx(char* y, int n, int m, int shifts[], int indx[]) {
    int j = 0;
    int limit = n - m;
 
    while (j <= limit ) {
        j += shifts[ y[j + m] ];
        indx[j] = j;
    }
}

void display_results(int n, int  res[]) {
    int j=0;
    for( int i =0; i < n; ++i )
        if ( res[i] == 1 ) {
		      j++;
//        printf("%d\n",i);
		}

//       printf("%d\n",j);
}

int main(int argc, char* argv[]) {
    int cuda_device = 0;
    size_t n = 0;
    size_t m = 0;

    if ( argc < 4 ) {
//        printf("Usage: ./a.out <device number> <pattern> <data file>\n");
        return -1;
    }

    if( argc > 1 )
        cuda_device = atoi( argv[1] );

    char* mainString = readfile(argv[3]);
    char* subString = (char*) malloc( (strlen(argv[2])) * sizeof(char) );
    strcpy(subString, argv[2]);
    n = strlen(mainString);
    m = strlen(subString);

    int* results=(int*)malloc(n * sizeof(int));

    int* l_shifts = (int*)malloc( ASIZE * sizeof(int) );
    for( int i = 0; i < ASIZE; ++i )
        l_shifts[i] = m + 1;
    int* l_indx = (int*) malloc( n * sizeof(int) );
    for( int i = 0; i < n; ++i ) {
        l_indx[i] = -1;
 	results[i]=1;
	}
	l_indx[0]=0;

//    hipError_t error;
    hipEvent_t start_event, stop_event;
    float time1, time2;
    
    checkCudaErrors( hipEventCreate(&start_event) );
	checkCudaErrors( hipEventCreate(&stop_event) );

    int num_devices=0;
    checkCudaErrors( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
 //       printf("Your system does not have a CUDA capable device\n");
        return 1;
    }
/*
    if( cuda_device >= num_devices )
    {
		if(num_devices==0)
//			printf("You have only 1 device and it's id is 0\n");
		else    
//		    printf("choose device ID between 0 and %d\n", num_devices-1);
        return 1;
    }
*/
    hipDeviceProp_t deviceProp;

    checkCudaErrors( hipGetDeviceProperties(&deviceProp, cuda_device) );
//    if( (1 == deviceProp.major) && (deviceProp.minor < 1))
//    printf("%s does not have compute capability 1.1 or later\n", deviceProp.name);

//    printf("Device name : %s\n", deviceProp.name );
//    printf("CUDA Capable SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
 //   printf("array_size   = %zd\n", n);

    char* d_substr = 0;
    int* d_shifts = 0;
    int* d_indx = 0;
    char* d_text = 0;
    int* d_results = 0;



    checkCudaErrors( hipMalloc((void**)&d_shifts, sizeof(int)*ASIZE));
    checkCudaErrors( hipMalloc((void**)&d_indx, n * sizeof(int)) );
    checkCudaErrors( hipMalloc((void**)&d_results, n * sizeof(int)) );
    checkCudaErrors( hipMalloc((void**)&d_substr, (m)*sizeof(char)) );
    checkCudaErrors( hipMalloc((void**)&d_text, (strlen(mainString))*sizeof(char)) );
	checkCudaErrors( hipMemcpy(d_shifts, l_shifts, sizeof(int) * ASIZE, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy(d_results, results, sizeof(int) * n, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy(d_text, mainString, sizeof(char)*(strlen(mainString)), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy(d_substr, subString, sizeof(char)*(m), hipMemcpyHostToDevice) );

 //   error = hipGetLastError();
 //   printf("%s\n", hipGetErrorString(error));

    dim3 threadsPerBlocks(ASIZE, 1);
    int t = m / threadsPerBlocks.x;
    int t1 = m % threadsPerBlocks.x;
    if ( t1 != 0 ) t += 1;
    dim3 numBlocks(t,1);

//    printf("Launching kernel with blocks=%d, threadsperblock=%d\n", numBlocks.x, threadsPerBlocks.x);

	hipEventRecord(start_event, 0);
		    processPattern<<<numBlocks,threadsPerBlocks>>>(d_substr, m, d_shifts);
		    hipDeviceSynchronize();
    hipEventRecord(stop_event, 0);
    hipEventSynchronize( stop_event );
    hipEventElapsedTime( &time1, start_event, stop_event );

	checkCudaErrors( hipMemcpy(l_shifts, d_shifts, sizeof(int) * ASIZE, hipMemcpyDeviceToHost ) );
    precomputeShiftIndx(mainString , n, m, l_shifts, l_indx);
    checkCudaErrors( hipMemcpy(d_indx, l_indx, n * sizeof(int), hipMemcpyHostToDevice) );

/*
//	For debugging
    for( int i = 0; i < ASIZE; ++i )
	printf("%d\t",l_shifts[i]);

	printf("\n\n");

    for( int i = 0; i < n; ++i )
	printf("%d\t",l_indx[i]);

	printf("\n\n");
	printf("%zd\t%zd",n,m);

	printf("\n\n");
*/

    t = n / threadsPerBlocks.x;
    t1 = n % threadsPerBlocks.x;
    if ( t1 != 0 ) t += 1;
    dim3 numBlocks2(t, 1);
 //   printf("Launching kernel with blocks=%d, threadsperblock=%d\n", numBlocks2.x, threadsPerBlocks.x);
    hipEventRecord(start_event, 0);
	    search<<<numBlocks2,threadsPerBlocks>>>(d_substr, m, d_text, n, d_shifts, d_indx, d_results);
    hipDeviceSynchronize();
    hipEventRecord(stop_event, 0);
    hipEventSynchronize( stop_event );
    hipEventElapsedTime( &time2, start_event, stop_event );

    hipEventDestroy( start_event );
    hipEventDestroy( stop_event );
 //   printf("%f+%f=%f milliseconds\t",time1, time2, time1+time2);
   printf("%f\t",time1+time2);
    checkCudaErrors( hipMemcpy(results, d_results, n * sizeof(int), hipMemcpyDeviceToHost) );

    display_results(n, results);


	hipFree(d_substr);
	hipFree(d_shifts);
	hipFree(d_indx);
	hipFree(d_text);
	hipFree(d_results);
//	free(mainString);
	free(subString);
	free(l_indx);
	free(l_shifts);
	free(results);
	
	hipDeviceReset();
}
