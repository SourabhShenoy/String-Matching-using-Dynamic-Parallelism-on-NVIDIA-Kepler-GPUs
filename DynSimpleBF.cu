#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <fstream>
# include <sys/time.h>
#define ASIZE 256
struct timeval tim;
double dTime1; 

__global__ void compare(int idx,char* x, char* y,int results[]) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(x[id]!=y[idx+id]) {
		results[idx]=0;
		return;
	} else {
		return;
	}
}


__global__ void search(char *x, int m, char* y, int n, int results[]) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
     
    if ( idx > (n - m) ) {results[idx]=0; return;}

	   if(x[0]==y[idx] && x[m-1]==y[idx+m-1]) {
	if(m>2)
		compare<<<1,m>>>(idx,x,y,results);
		return;
	  } else {
		results[idx]=0;
		return;
	} 
}

char* readfile(const char* filename) {
	FILE* f;
	char* data;
	f= fopen(filename, "r");
	if ( f != NULL )  {
		fseek(f,0,SEEK_END);
	    int size=ftell(f);
		fseek(f,0,SEEK_SET);
		data = (char*)malloc((size+1) * sizeof(char));
		fread(data, size,1,f);
	}
	fclose(f);
	return data;
}

void display_results(int n, int  res[]) {
	int c=0;
    for( int i =0; i < n; ++i )
        if ( res[i] == 1 )
		c++;
//		printf("\n\nCount:%d\n\n",c);
//            printf("%d. Found match at %d\n",j++, i);

}

int main(int argc, char* argv[]) {
    int cuda_device = 0;
    size_t n = 0;
    size_t m = 0;

    if ( argc < 4 ) {
  //      printf("Usage: ./a.out <device number> <pattern> <data file>\n");
        return -1;
    }

    if( argc > 1 )
        cuda_device = atoi( argv[1] );

	char* mainString = readfile(argv[3]);
    char* subString = (char*) malloc( (strlen(argv[2])) * sizeof(char) );
    strcpy(subString, argv[2]);
    n = strlen(mainString);
    m = strlen(subString);

	int* results=(int*)malloc(n * sizeof(int));
	for( int i = 0; i < n; ++i ) {
		results[i]=1;
	}

  //  hipError_t error;
    hipEvent_t start_event, stop_event;
    float time1;
    
    checkCudaErrors( hipEventCreate(&start_event) );
	checkCudaErrors( hipEventCreate(&stop_event) );

    int num_devices=0;
    checkCudaErrors( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
  //      printf("Your system does not have a CUDA capable device\n");
        return 1;
    }
/*
    if( cuda_device >= num_devices )
    {
		if(num_devices==0)
//			printf("You have only 1 device and it's id is 0\n");
		else    
//		    printf("choose device ID between 0 and %d\n", num_devices-1);
        return 1;
    }
*/
    //hipSetDevice( cuda_device );
    hipDeviceProp_t deviceProp;

    checkCudaErrors( hipGetDeviceProperties(&deviceProp, cuda_device) );
 //   if( (1 == deviceProp.major) && (deviceProp.minor < 1))
//   printf("%s does not have compute capability 1.1 or later\n", deviceProp.name);

//    printf("Device name : %s\n", deviceProp.name );
//    printf("CUDA Capable SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
 //   printf("array_size   = %zd\n", n);

    char* d_substr = 0;
    char* d_text = 0;
    int* d_results = 0;

    checkCudaErrors( hipMalloc((void**)&d_results, n * sizeof(int)) );
    checkCudaErrors( hipMalloc((void**)&d_substr, (m)*sizeof(char)) );
    checkCudaErrors( hipMalloc((void**)&d_text, (strlen(mainString))*sizeof(char)) );
    checkCudaErrors( hipMemcpy(d_results, results, sizeof(int) * n, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy(d_text, mainString, sizeof(char)*(strlen(mainString)), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy(d_substr, subString, sizeof(char)*(m), hipMemcpyHostToDevice) );
//    error = hipGetLastError();
 //   printf("%s\n", hipGetErrorString(error));

    dim3 threadsPerBlocks(ASIZE, 1);
    int t = n / threadsPerBlocks.x;
    int t1 = n % threadsPerBlocks.x;
    if ( t1 != 0 ) t += 1;
    dim3 numBlocks(t,1);

//    printf("Launching kernel with blocks=%d, threadsperblock=%d\n", numBlocks.x, threadsPerBlocks.x);

    hipEventRecord(start_event, 0);
		    search<<<numBlocks,threadsPerBlocks>>>(d_substr, m, d_text, n, d_results);
    hipDeviceSynchronize();
    hipEventRecord(stop_event, 0);
    hipEventSynchronize( stop_event );
    hipEventElapsedTime( &time1, start_event, stop_event );

    hipEventDestroy( start_event );
    hipEventDestroy( stop_event );

    printf("%lf\t",time1);

    checkCudaErrors( hipMemcpy(results, d_results, n * sizeof(int), hipMemcpyDeviceToHost) );
	display_results(n, results);

	hipFree(d_substr);
	hipFree(d_text);
	hipFree(d_results);
	free(mainString);
	free(subString);
	free(results);
	
	hipDeviceReset();
}
