#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <string.h>


#define cutilSafeCall(x) checkCudaErrors(x)
#define cutilCheckMsg(x) getLastCudaError(x)

#define BLOCK_SIZE 32

static __constant__ char d_stringPattern[BLOCK_SIZE];

__global__ void compare(int idx,const char* x, char* y,int results[],int shift) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
//			printf("%d %c %c\n",id,x[id],y[id+idx]);
	if(x[id]!=y[idx+id]) {
		results[shift]=0;
	}
}

template<class TDATA, unsigned int SUBSTRLEN, unsigned int LEN>
__global__ void strmatch(const char* substr, const char* data,  size_t len, size_t substrlen, int* results) {
    __shared__ char sharedData[BLOCK_SIZE + SUBSTRLEN];
 
    int shft = blockIdx.x * blockDim.x + threadIdx.x;

	if ( threadIdx.x == (warpSize - 1) )
		for(int i = 0; i < SUBSTRLEN; ++i)
			sharedData[threadIdx.x + i] = data[shft+i];
	else 
		sharedData[threadIdx.x] = data[shft];
    __syncthreads();

    const char* s2 = substr;

/*    if ( (LEN - shft) < SUBSTRLEN ) {
        results[shft] = 0;
        return;
    }
*/
		char *c = sharedData;
//			strcpy(c,sharedData);

	//		printf("%d %d \n",shft,results[shft]);

		if((s2[0] == sharedData[threadIdx.x]) && (s2[SUBSTRLEN-1] == sharedData[threadIdx.x + SUBSTRLEN - 1])) {
					compare<<<1,SUBSTRLEN>>>(threadIdx.x,s2,c,results,shft);
//			printf("%d %s %s %d %d\n",threadIdx.x,s2,c,results[shft],shft);
		} else {
				results[shft]=0;
		}
	//			printf("%d\n",results[shft]);

/*
    for( int i = threadIdx.x ; curr_marker <= SUBSTRLEN && i < LEN; curr_marker++, i++ ) {
        if ( s2[curr_marker] && (s2[curr_marker] != sharedData[i]) ) {
            yes = 0;
            break;
        }
    }
*/
}

char* readfile(const char* filename) {
	FILE* f;
	char* data;
	f= fopen(filename, "r");
	if ( f != NULL )  {
		fseek(f,0,SEEK_END);
	    int size=ftell(f);
		fseek(f,0,SEEK_SET);
		data = (char*)malloc((size+1) * sizeof(char));
		fread(data, size,1,f);
	}
	fclose(f);
	return data;
}

//
// simple print function to see the shifts in the res array
//
void print_shifts(int *iptr, int strlen) {
int j=0;
    for(unsigned int i = 0; i < strlen; i++ ) {
  //      printf("%d\n",iptr[i]);
        if (iptr[i] == 1) {
		j++;
//        printf("%d\n",i);
	}
}
		printf("\n\nCount:%d\n\n",j);    
//        printf("Match found at position: %d\n", i);
   
}

int main(int argc, char** argv) {
    int cuda_device = 0; // variable used to denote the device ID
    int n = 0;           // number of ints in the data set
    hipError_t error;   // capture returned error code
    hipEvent_t start_event, stop_event; // data structures to capture events in GPU
    float time;
	// Sanity checks
	{
	    // check the compute capability of the device
	    int num_devices=0;
	    cutilSafeCall( hipGetDeviceCount(&num_devices) );
	    if(0==num_devices)
	    {
//	        printf("your system does not have a CUDA capable device\n");
	        return 1;
	    }
    	if( argc > 1 )
       		cuda_device = atoi( argv[1] );

	    // check if the command-line chosen device ID is within range, exit if not
	    if( cuda_device >= num_devices )
	    {
//	        printf("choose device ID between 0 and %d\n", num_devices-1);
	        return 1;
	    }

    	hipSetDevice( cuda_device );

		if ( argc < 4 ) {
//      		printf("Usage: bruteforcematcher <device number> <pattern> <data file>\n");
      		return -1;
    	}
	} // end of sanity checks


    // read in the filename and string pattern to be searched
    char* mainString = readfile( argv[3] );
    char* subString = (char*)malloc( (strlen(argv[2])) * sizeof(char) );
    strcpy(subString, argv[2]);
    n = strlen(mainString);
//    m = strlen(subString);

    // initializing the GPU timers
    cutilSafeCall( hipEventCreate(&start_event) );
    cutilSafeCall( hipEventCreate(&stop_event) );
   
    hipDeviceProp_t deviceProp;
    cutilSafeCall( hipGetDeviceProperties(&deviceProp, cuda_device) );
/*    if( (1 == deviceProp.major) && (deviceProp.minor < 1))
        printf("%s does not have compute capability 1.1 or later\n", deviceProp.name);

    printf("> Device name : %s\n", deviceProp.name );
    printf("> CUDA Capable SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
    printf("> Data Size = %d\n", n);
    printf("> String Pattern = %s\n\n", subString);
*/
    // allocate host memory
    char* d_substr = 0;
    char* d_data = 0;
    int*  d_finalres = 0;
    int* finalres = (int*)malloc( (strlen(mainString))*sizeof(int) );
		
		for(int i=0;i<n;i++) finalres[i]=1;

    cutilSafeCall( hipMalloc((void**)&d_substr, (strlen(subString))*sizeof(char)) );
    cutilSafeCall( hipMalloc((void**)&d_finalres, (strlen(mainString))*sizeof(int)) );
    cutilSafeCall( hipMemcpy(d_finalres, finalres, sizeof(int)*(n), hipMemcpyHostToDevice ) );
    
cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_stringPattern), subString, sizeof(char)*(strlen(subString))) );
    cutilSafeCall( hipMalloc((void**)&d_data, (strlen(mainString))*sizeof(char)) );

    cutilSafeCall( hipMemcpy(d_data, mainString, sizeof(char)*(strlen(mainString)), hipMemcpyHostToDevice ) );
    cutilSafeCall( hipMemcpy(d_substr, subString, sizeof(char)*(strlen(subString)), hipMemcpyHostToDevice) );
//		cutilSafeCall( hipMemset(d_finalres, 1, sizeof(int)*strlen(mainString)) );
    
    dim3 threadsPerBlocks(BLOCK_SIZE, 1);
    dim3 numBlocks((int)ceil((float)n/threadsPerBlocks.x), 1);

//	printf("Launching kernel with %d blocks, %d threads per block\n", numBlocks.x, threadsPerBlocks.x);
	// start timer!
    hipEventRecord(start_event, 0);

	// conduct actual search!!
//	    strmatch<char*, m, n><<<numBlocks,threadsPerBlocks>>>(d_substr, d_data, strlen(mainString), strlen(subString), d_finalres);
	    strmatch<char*, 5, 1181741><<<numBlocks,threadsPerBlocks>>>(d_substr, d_data, strlen(mainString), strlen(subString), d_finalres);
	// stop timer
    hipEventRecord(stop_event, 0);
    hipEventSynchronize( stop_event );

    hipEventElapsedTime( &time, start_event, stop_event );
    hipEventDestroy( start_event ); // cleanup
    hipEventDestroy( stop_event ); // cleanup

    error = hipGetLastError();
    if ( error ) {
//    	printf("Error caught: %s\n", hipGetErrorString( error ));
    }
    printf("%f\t", time);

    cutilSafeCall( hipMemcpy(finalres, d_finalres, (strlen(mainString))*sizeof(int), hipMemcpyDeviceToHost) );

 
    // check whether the output is correct
//    printf("-------------------------------\n");
    print_shifts(finalres, strlen(mainString)+1);
//    printf("-------------------------------\n");

    hipFree(d_substr);
    hipFree(d_data);
    hipFree(d_finalres);
    free(finalres);
	free(subString);
	free(mainString);
	
    return 0;
}
