
#include <hip/hip_runtime.h>
__global__ void reverseArrayBlock(int *d_out, int *d_in)
{
    int in = threadIdx.x;
    int out = blockDim.x - 1 - threadIdx.x;
    d_out[out] = d_in[in];
}
