
#include <hip/hip_runtime.h>

// Part3: implement the kernel
__global__ void reverseArrayBlock(int *d_out, int *d_in)
{
    int inOffset  = blockDim.x * blockIdx.x;
    int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
    int in  = inOffset + threadIdx.x;
    int out = outOffset + (blockDim.x - 1 - threadIdx.x);
    d_out[out] = d_in[in];
}

