#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

// Part 3 of 5: implement kernel 
// It receives an array A as argument
// Each thread finds its linearized id in the grid and sets A[id] = id
__global__ void myFirstKernel(                           )
{



}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc(                                              );

    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid(             );
    dim3 dimBlock(             );
    myFirstKernel<<<             ,             >>>(             );

    // Part 4 of 5: device to host copy
    hipMemcpy(                                                 );

    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i <              ; i++)
            assert(h_a[i] ==      );
    }


    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    printf("Correct!\n");

    return 0;
}
