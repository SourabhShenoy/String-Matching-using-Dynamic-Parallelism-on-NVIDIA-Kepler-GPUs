#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

#include "timer.h"

/*
 * **CUDA KERNEL** 
 * 
 * Compute the sum of two vectors 
 *   C[i] = A[i] + B[i]
 * 
 */
__global__ void vecAdd(float* a, float* b, float* c) {

 /* Calculate index for this thread */
 int i = blockIdx.x * blockDim.x + threadIdx.x;

 /* Compute the element of C */
 c[i] = a[i] + b[i];
}

void compute_vec_add(int N, float *a, float* b, float *c); 

/*
 * 
 * Host code to drive the CUDA Kernel
 * 
 */
int main() { 

float *d_a, *d_b, *d_c;
float *h_a, *h_b, *h_c, *h_temp;
int i; 
int N = 1024 * 1;

//struct stopwatch_t* timer = NULL;
long double t_pcie_htd, t_pcie_dth, t_kernel, t_cpu;

/* Setup timers */
//stopwatch_init ();
//timer = stopwatch_create ();

/*
  Create the vectors
*/
h_a = (float *) malloc(sizeof(float) * N);
h_b = (float *) malloc(sizeof(float) * N);
h_c = (float *) malloc(sizeof(float) * N);

/*
  Set the initial values of h_a, h_b, and h_c
*/
for (i=0; i < N; i++) {
	h_a[i] = (float) (rand() % 100) / 10.0;
	h_b[i] = (float) (rand() % 100) / 10.0;
	h_c[i] = 0.0;
}


/*
  Allocate space on the GPU
*/
CUDA_CHECK_ERROR(hipMalloc(&d_a, sizeof(float) * N));
CUDA_CHECK_ERROR(hipMalloc(&d_b, sizeof(float) * N));
CUDA_CHECK_ERROR(hipMalloc(&d_c, sizeof(float) * N));

/*
  Copy d_a and d_b from CPU to GPU
*/
//stopwatch_start (timer);
CUDA_CHECK_ERROR(hipMemcpy(d_a, h_a, sizeof(float) * N, hipMemcpyHostToDevice));
CUDA_CHECK_ERROR(hipMemcpy(d_b, h_b, sizeof(float) * N, hipMemcpyHostToDevice));
//t_pcie_htd = stopwatch_stop (timer);
printf ("Time to transfer data from host to device: %Lg secs\n", 
				 t_pcie_htd);

/*
  Run N/256 blocks of 256 threads each
*/
dim3 GS (N/256, 1, 1);
dim3 BS (256, 1, 1);

//stopwatch_start (timer);
vecAdd<<<GS, BS>>>(d_a, d_b, d_c);
hipDeviceSynchronize ();
//t_kernel = stopwatch_stop (timer);
printf ("Time to execute GPU kernel: %Lg secs\n", 
				 t_kernel);

/*
  Copy d_cfrom GPU to CPU
*/
//stopwatch_start (timer);
CUDA_CHECK_ERROR(hipMemcpy(h_c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost));
//t_pcie_dth = stopwatch_stop (timer);
printf ("Time to transfer data from device to host: %Lg secs\n", 
				 t_pcie_dth);


/* 
	Double check errors
 */
h_temp = (float *) malloc(sizeof(float) * N);
//stopwatch_start (timer);
compute_vec_add (N, h_a, h_b, h_temp);
//t_cpu = stopwatch_stop (timer);
printf ("Time to execute CPU program: %Lg secs\n", 
				 t_cpu);

int cnt = 0;
for(int i = 0; i < N; i++) {
	if(abs(h_temp[i] - h_c[i]) > 1e-5) cnt++;
}
printf("number of errors: %d out of %d\n", cnt, N);


/*
 Free the device memory
*/
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);


/*
 Free the host memory
*/
free(h_a);
free(h_b);
free(h_c);

/* 
 Free timer 
*/
//stopwatch_destroy (timer);

if(cnt == 0) {
	printf("\n\nSuccess\n");
}
}

void
compute_vec_add(int N, float *a, float* b, float *c) {
int i;
for (i=0;i<N;i++)
c[i]=a[i]+b[i];
}


