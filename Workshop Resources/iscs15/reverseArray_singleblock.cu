#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

// Part 1 of 1: implement kernel to reverse elements in argument array
__global__ void reverseArrayBlock( )
{




}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 256;

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numBlocks = 1;
    int numThreadsPerBlock = dimA;

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock >>>( d_b, d_a );

    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i );
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    printf("Correct!\n");

    return 0;
}

